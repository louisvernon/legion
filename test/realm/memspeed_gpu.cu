#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
//include <cuda_runtime.h>

extern "C" {
  double gpu_seqwr_test(void *buffer, size_t reps, size_t elements);
  double gpu_seqrd_test(void *buffer, size_t reps, size_t elements);
  double gpu_rndwr_test(void *buffer, size_t reps, size_t elements);
  double gpu_rndrd_test(void *buffer, size_t reps, size_t elements);
  double gpu_latency_test(void *buffer, size_t reps, size_t elements);
}

__global__ void gpu_seqwr_kernel(int *buffer, size_t reps, size_t elements)
{
  for(size_t j = 0; j < reps; j++) {
    size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;
    while(ofs < elements) {
      buffer[ofs] = 0;
      ofs += step;
    }
  }
}

__global__ void gpu_seqrd_kernel(int *buffer, size_t reps, size_t elements)
{
  int errors = 0;
  for(size_t j = 0; j < reps; j++) {
    size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;
    while(ofs < elements) {
      int val = buffer[ofs];
      if(val != 0)
	errors++;
      ofs += step;
    }
  }
  if(errors > 0)
    buffer[0] = errors;
}

__global__ void gpu_rndwr_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
  // we don't want completely random writes here, since the performance would be awful
  // instead, let each warp move around randomly, but keep the warp coalesced on 128B-aligned
  //  accesses
  for(size_t j = 0; j < reps; j++) {
    // starting point is naturally aligned
    size_t p = blockIdx.x * blockDim.x + threadIdx.x;
    // if we start outside the block, sit this out (just to keep small runs from crashing)
    if(p >= elements) break;

    // quadratic stepping via "acceleration" and "velocity"
    size_t a = 548191;
    size_t v = 24819 + (p >> 5);  // velocity has to be different for each warp

    for(size_t i = 0; i < steps; i++) {
      size_t prev = p;
      // delta is multiplied by 32 elements so warp stays converged (velocity is the
      //  same for all threads in the warp)
      p = (p + (v << 5)) % elements;
      v = (v + a) % elements;
      buffer[prev] = p;
    }
  }
}

__global__ void gpu_rndrd_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
  // we don't want completely random writes here, since the performance would be awful
  // instead, let each warp move around randomly, but keep the warp coalesced on 128B-aligned
  //  accesses
  int errors = 0;
  for(size_t j = 0; j < reps; j++) {
    // starting point is naturally aligned
    size_t p = blockIdx.x * blockDim.x + threadIdx.x;
    // if we start outside the block, sit this out (just to keep small runs from crashing)
    if(p >= elements) break;

    // quadratic stepping via "acceleration" and "velocity"
    size_t a = 548191;
    size_t v = 24819 + (p >> 5);  // velocity has to be different for each warp

    for(size_t i = 0; i < steps; i++) {
      size_t prev = p;
      // delta is multiplied by 32 elements so warp stays converged (velocity is the
      //  same for all threads in the warp)
      p = (p + (v << 5)) % elements;
      v = (v + a) % elements;

      if(buffer[prev] != p)
	errors++;
    }
  }
  if((errors > 0) && (reps > elements))
    buffer[0] = errors;
}

__global__ void gpu_latency_setup_kernel(int *buffer, size_t delta, size_t elements)
{
  size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
  size_t step = blockDim.x * gridDim.x;
  while(ofs < elements) {
    size_t tgt = ofs + delta;
    if(tgt > elements)
      tgt -= elements;
    buffer[ofs] = tgt;
    ofs += step;
  }
}

__global__ void gpu_latency_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
  int errors = 0;
  // this is done with just a single thread
  for(size_t j = 0; j < reps; j++) {
    int p = j & 31;

    for(size_t i = 0; i < steps; i++) {
      int next = buffer[p];

      if((next >= 0) && (next < elements)) {
	p = next;
      } else {
	printf("%d -> %d\n", p, next);
	p = 0;
	errors++;
      }
    }
  }
  if((errors > 0) && (reps > elements))
    buffer[0] = errors;
}	  

static void get_launch_params(int *grid_size, int *block_size)
{
 // want to fill the GPU precisely, so figure out how many threads we can fit
  //  (our register count should be low enough to not be limited by RF size)
  int device;
  struct hipDeviceProp_t props;
  hipError_t ret;

  ret = hipGetDevice(&device);
  assert(ret == hipSuccess);

  ret = hipGetDeviceProperties(&props, device);
  assert(ret == hipSuccess);

  // seems like this should be queryable?
  int ctas_per_sm = 8;
  int threads_per_sm = 512;
  int threads_per_cta = threads_per_sm / ctas_per_sm;
  int total_ctas = props.multiProcessorCount * ctas_per_sm;

  *grid_size = total_ctas;
  *block_size = threads_per_cta;
}

double gpu_seqwr_test(void *buffer, size_t reps, size_t elements)
{
  int grid_size, block_size;
  get_launch_params(&grid_size, &block_size);

  hipEvent_t t_start, t_end;
  hipEventCreate(&t_start);
  hipEventCreate(&t_end);
  hipEventRecord(t_start, 0);
  gpu_seqwr_kernel<<< grid_size, block_size >>>((int *)buffer, reps, elements);
  hipEventRecord(t_end, 0);
  
  hipError_t ret = hipEventSynchronize(t_end);
  assert(ret == hipSuccess);

  float elapsed;
  hipEventElapsedTime(&elapsed, t_start, t_end);

  hipEventDestroy(t_start);
  hipEventDestroy(t_end);

  // BW units are GB/s (a.k.a. B/ns) - elapsed is in ms
  double seqwr_bw = 1e-6 * reps * elements * sizeof(int) / elapsed;
  return seqwr_bw;
}

double gpu_seqrd_test(void *buffer, size_t reps, size_t elements)
{
  int grid_size, block_size;
  get_launch_params(&grid_size, &block_size);

  hipEvent_t t_start, t_end;
  hipEventCreate(&t_start);
  hipEventCreate(&t_end);
  hipEventRecord(t_start, 0);
  gpu_seqrd_kernel<<< grid_size, block_size >>>((int *)buffer, reps, elements);
  hipEventRecord(t_end, 0);
  
  hipError_t ret = hipEventSynchronize(t_end);
  assert(ret == hipSuccess);

  float elapsed;
  hipEventElapsedTime(&elapsed, t_start, t_end);

  hipEventDestroy(t_start);
  hipEventDestroy(t_end);

  // BW units are GB/s (a.k.a. B/ns) - elapsed is in ms
  double seqrd_bw = 1e-6 * reps * elements * sizeof(int) / elapsed;
  return seqrd_bw;
}

double gpu_rndwr_test(void *buffer, size_t reps, size_t elements)
{
  int grid_size, block_size;
  get_launch_params(&grid_size, &block_size);
  int total_threads = grid_size * block_size;

  size_t steps = 64;

  hipEvent_t t_start, t_end;
  hipEventCreate(&t_start);
  hipEventCreate(&t_end);
  hipEventRecord(t_start, 0);
  gpu_rndwr_kernel<<< grid_size, block_size >>>((int *)buffer, reps, steps, elements);
  hipEventRecord(t_end, 0);
  
  hipError_t ret = hipEventSynchronize(t_end);
  assert(ret == hipSuccess);

  float elapsed;
  hipEventElapsedTime(&elapsed, t_start, t_end);

  hipEventDestroy(t_start);
  hipEventDestroy(t_end);

  // BW units are GB/s (a.k.a. B/ns) - elapsed is in ms
  double rndwr_bw = 1e-6 * reps * total_threads * steps * sizeof(int) / elapsed;
  return rndwr_bw;
}

double gpu_rndrd_test(void *buffer, size_t reps, size_t elements)
{
  int grid_size, block_size;
  get_launch_params(&grid_size, &block_size);
  int total_threads = grid_size * block_size;

  size_t steps = 64;

  hipEvent_t t_start, t_end;
  hipEventCreate(&t_start);
  hipEventCreate(&t_end);
  hipEventRecord(t_start, 0);
  gpu_rndrd_kernel<<< grid_size, block_size >>>((int *)buffer, reps, steps, elements);
  hipEventRecord(t_end, 0);
  
  hipError_t ret = hipEventSynchronize(t_end);
  assert(ret == hipSuccess);

  float elapsed;
  hipEventElapsedTime(&elapsed, t_start, t_end);

  hipEventDestroy(t_start);
  hipEventDestroy(t_end);

  // BW units are GB/s (a.k.a. B/ns) - elapsed is in ms
  double rndrd_bw = 1e-6 * reps * total_threads * steps * sizeof(int) / elapsed;
  return rndrd_bw;
}

double gpu_latency_test(void *buffer, size_t reps, size_t elements)
{
  int grid_size, block_size;
  get_launch_params(&grid_size, &block_size);

  size_t steps = 1024;

  // initialize the data with something that makes large jumps through memory
  // for now, assume that no prefetcher will take notice of the stride
  // try to pick something that won't get too close to a multiple of the element size
  //  in 'steps' tries
  size_t delta = (((steps >> 2) - 3) / (steps + 1.0)) * elements;
  if(delta == 0) delta = 1;
  gpu_latency_setup_kernel<<< grid_size, block_size >>>((int *)buffer, delta, elements);

  hipEvent_t t_start, t_end;
  hipEventCreate(&t_start);
  hipEventCreate(&t_end);
  hipEventRecord(t_start, 0);
  gpu_latency_kernel<<< 1, 1 >>>((int *)buffer, reps, steps, elements);
  hipEventRecord(t_end, 0);
  
  hipError_t ret = hipEventSynchronize(t_end);
  assert(ret == hipSuccess);

  float elapsed;
  hipEventElapsedTime(&elapsed, t_start, t_end);

  hipEventDestroy(t_start);
  hipEventDestroy(t_end);

  // latency units are in ns - elapsed is in ms
  double latency = (elapsed * 1e6) / (reps * steps);
  return latency;
}